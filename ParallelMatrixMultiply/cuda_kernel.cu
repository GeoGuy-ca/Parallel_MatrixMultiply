#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <algorithm>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <Windows.h>

__global__ void multiplyKernel(unsigned long long *c, const unsigned long long *a, const unsigned long long *b, unsigned int m, unsigned int n, unsigned int o)
{
	int me = (blockIdx.y * blockDim.y) + (blockIdx.x * blockDim.x) + threadIdx.x;
	int col = me / m;
	int row = me % m;
	for (int element = 0; element < n; element++) {
		c[me] += a[element*m + row] * b[col*n + element];
	}
}

void MatrixMultiplyWithCuda(unsigned long long *c, const unsigned long long *a, const unsigned long long *b, unsigned int m, unsigned int n, unsigned int o)
{
	unsigned long long *dev_a = 0;
	unsigned long long *dev_b = 0;
	unsigned long long *dev_c = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_c, m * o * sizeof(long long));
	hipMalloc((void**)&dev_a, m * n * sizeof(long long));
	hipMalloc((void**)&dev_b, n * o * sizeof(long long));

	hipMemcpy(dev_a, a, m * n * sizeof(long long), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * o * sizeof(long long), hipMemcpyHostToDevice);

	dim3 threads(1024);//max 1024 threads per block!
	dim3 blocks(m / threads.x + 1, o / threads.x + 1);
	multiplyKernel<<<blocks, threads>>>(dev_c, dev_a, dev_b, m, n, o);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, m * o * sizeof(long long), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}


void MatrixRandomFill(unsigned long long *matrix, unsigned int m, unsigned int n)
{
	unsigned long long *dev_matrix = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_matrix, m * n * sizeof(long long));

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
	hiprandSetGeneratorOffset(gen, GetTickCount64());
	hiprandSetQuasiRandomGeneratorDimensions(gen, m * n);
	hiprandGenerateLongLong(gen, dev_matrix, m * n);
	
	hipDeviceSynchronize();
	hipMemcpy(matrix, dev_matrix, m * n * sizeof(long long), hipMemcpyDeviceToHost);

	hipFree(dev_matrix);
}